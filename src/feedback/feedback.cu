
#include <hip/hip_runtime.h>
#if defined(FEEDBACK) && defined(PARTICLES_GPU) && defined(PARTICLE_AGE) && defined(PARTICLE_IDS)

  #include <math.h>
  #include <stdio.h>
  #include <stdlib.h>
  #include <unistd.h>

  #include <cstring>
  #include <fstream>
  #include <sstream>
  #include <vector>

  #include "../global/global.h"
  #include "../global/global_cuda.h"
  #include "../grid/grid3D.h"
  #include "../io/io.h"
  #include "feedback.h"

  #define FEED_INFO_N     8
  #define i_RES           1
  #define i_UNRES         2
  #define i_ENERGY        3
  #define i_MOMENTUM      4
  #define i_UNRES_ENERGY  5
  #define i_WIND_MOMENTUM 6
  #define i_WIND_ENERGY   7

  // the starburst 99 total stellar mass input
  // stellar wind momentum fluxes and SN rates
  // must be divided by this to get per solar
  // mass values.
  #define S_99_TOTAL_MASS 1e6

  #define TPB_FEEDBACK 128
  // seed for poisson random number generator
  #define FEEDBACK_SEED 42

namespace feedback
{
Real *dev_snr, snr_dt, time_sn_start, time_sn_end;
Real *dev_sw_p, *dev_sw_e, sw_dt, time_sw_start, time_sw_end;
int snr_n;
}  // namespace feedback

  #ifndef O_HIP
inline __device__ double atomicMax(double* address, double val)
{
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old             = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}
  #endif  // O_HIP

inline __device__ Real Calc_Timestep(Real gamma, Real* density, Real* momentum_x, Real* momentum_y, Real* momentum_z,
                                     Real* energy, int index, Real dx, Real dy, Real dz)
{
  Real dens  = fmax(density[index], DENS_FLOOR);
  Real d_inv = 1.0 / dens;
  Real vx    = momentum_x[index] * d_inv;
  Real vy    = momentum_y[index] * d_inv;
  Real vz    = momentum_z[index] * d_inv;
  Real P     = fmax((energy[index] - 0.5 * dens * (vx * vx + vy * vy + vz * vz)) * (gamma - 1.0), TINY_NUMBER);
  Real cs    = sqrt(gamma * P * d_inv);
  return fmax(fmax((fabs(vx) + cs) / dx, (fabs(vy) + cs) / dy), (fabs(vz) + cs) / dz);
}

/** the prescription for dividing a scalar quantity between 3x3x3 cells is done
   by imagining a 2x2x2 cell volume around the SN.  These fractions, then,
   represent the linear extent of this volume into the cell in question. For i=0
   this should be 1*1/2. For i=-1 this should be (1-dx)*1/2. For i=+1 this
   should be dx*1/2. In the above the 1/2 factor is normalize over 2
   cells/direction.
  */
inline __device__ Real Frac(int i, Real dx) { return (-0.5 * i * i - 0.5 * i + 1 + i * dx) * 0.5; }

inline __device__ Real D_Frac(int i, Real dx)
{
  return (dx > 0.5) * i * (1 - 2 * dx) + ((i + 1) * dx + 0.5 * (i - 1)) - 3 * (i - 1) * (i + 1) * (0.5 - dx);
}

/** This function used for debugging potential race conditions.  Feedback from neighboring
    particles could simultaneously alter one hydro cell's conserved quantities.
 */
inline __device__ bool Particle_Is_Alone(Real* pos_x_dev, Real* pos_y_dev, Real* pos_z_dev, part_int_t n_local,
                                         int gtid, Real dx)
{
  Real x0 = pos_x_dev[gtid];
  Real y0 = pos_y_dev[gtid];
  Real z0 = pos_z_dev[gtid];
  // Brute force loop to see if particle is alone
  for (int i = 0; i < n_local; i++) {
    if (i == gtid) continue;
    if (abs(x0 - pos_x_dev[i]) > dx) continue;
    if (abs(y0 - pos_y_dev[i]) > dx) continue;
    if (abs(z0 - pos_z_dev[i]) > dx) continue;
    // If we made it here, something is too close.
    return false;
  }
  return true;
}

inline __device__ Real Get_Average_Density(Real* density, int xi, int yi, int zi, int nx_grid, int ny_grid, int n_ghost)
{
  Real d_average = 0.0;
  for (int i = -1; i < 2; i++) {
    for (int j = -1; j < 2; j++) {
      for (int k = -1; k < 2; k++) {
        d_average +=
            density[(xi + n_ghost + i) + (yi + n_ghost + j) * nx_grid + (zi + n_ghost + k) * nx_grid * ny_grid];
      }
    }
  }
  return d_average / 27;
}

inline __device__ Real Get_Average_Number_Density_CGS(Real* density, int xi, int yi, int zi, int nx_grid, int ny_grid,
                                                      int n_ghost)
{
  return Get_Average_Density(density, xi, yi, zi, nx_grid, ny_grid, n_ghost) * DENSITY_UNIT / (MU * MP);
}

  #ifndef NO_SN_FEEDBACK
/**
 * @brief
 * -# Read in SN rate data from Starburst 99. If no file exists, assume a
 * constant rate.
 *
 * @param P pointer to parameters struct. Passes in starburst 99 filename and
 * random number gen seed.
 */
void feedback::Init_State(struct Parameters* P)
{
  chprintf("feedback::Init_State start\n");
  std::string snr_filename(P->snr_filename);
  if (not snr_filename.empty()) {
    chprintf("Specified a SNR filename %s.\n", snr_filename.data());

    // read in array of supernova rate values.
    std::ifstream snr_in(snr_filename);
    if (!snr_in.is_open()) {
      chprintf("ERROR: but couldn't read SNR file.\n");
      exit(-1);
    }

    std::vector<Real> snr_time;
    std::vector<Real> snr;

    const int N_HEADER    = 7;    // S'99 has 7 rows of header information
    const char* s99_delim = " ";  // S'99 data separator
    std::string line;
    int line_counter = 0;

    while (snr_in.good()) {
      std::getline(snr_in, line);
      if (line_counter++ < N_HEADER) continue;  // skip header processing

      int i      = 0;
      char* data = strtok(line.data(), s99_delim);
      while (data != nullptr) {
        if (i == 0) {
          // in the following divide by # years per kyr (1000)
          snr_time.push_back(std::stof(std::string(data)) / 1000);
        } else if (i == 1) {
          snr.push_back(pow(10, std::stof(std::string(data))) * 1000 / S_99_TOTAL_MASS);
        }

        if (i > 0) {
          break;  // only care about the first 2 items.  Once i = 1 can break
                  // here.
        }
        data = strtok(nullptr, s99_delim);
        i++;
      }
    }

    time_sn_end   = snr_time[snr_time.size() - 1];
    time_sn_start = snr_time[0];
    // the following is the time interval between data points
    // (i.e. assumes regular temporal spacing)
    snr_dt = (time_sn_end - time_sn_start) / (snr.size() - 1);

    GPU_Error_Check(cudaMalloc((void**)&dev_snr, snr.size() * sizeof(Real)));
    GPU_Error_Check(cudaMemcpy(dev_snr, snr.data(), snr.size() * sizeof(Real), cudaMemcpyHostToDevice));

  } else {
    chprintf("No SN rate file specified.  Using constant rate\n");
    time_sn_start = DEFAULT_SN_START;
    time_sn_end   = DEFAULT_SN_END;
  }
}
  #endif  // NO_SN_FEEDBACK

  #ifndef NO_WIND_FEEDBACK
/**
 * @brief
 * Read in Stellar wind data from Starburst 99. If no file exists, assume a
 * constant rate.
 *
 *
 * @param P pointer to parameters struct. Passes in starburst 99 filepath
 */
void feedback::Init_Wind_State(struct Parameters* P)
{
  chprintf("Init_Wind_State start\n");
  std::string sw_filename(P->sw_filename);
  if (sw_filename.empty()) {
    chprintf("must specify a stellar wind file.\n");
    exit(-1);
  }

  chprintf("Specified a stellar wind filename %s.\n", sw_filename.data());

  // read in array of supernova rate values.
  std::ifstream sw_in(sw_filename);
  if (!sw_in.is_open()) {
    chprintf("ERROR: couldn't read stellar wind file.\n");
    exit(-1);
  }

  std::vector<Real> sw_time;
  std::vector<Real> sw_p;
  std::vector<Real> sw_e;

  const int N_HEADER_LINES = 7;  // S'99 has 7 rows of header information
  const int COL_TIME       = 0;
  const int COL_POWER      = 1;
  const int COL_ALL_P_FLUX = 7;

  const char* s99_delim = " ";  // S'99 data separator
  std::string line;
  int line_counter = 0;

  while (sw_in.good()) {
    std::getline(sw_in, line);
    if (line_counter++ < N_HEADER_LINES) continue;  // skip header processing

    int i      = 0;
    char* data = strtok(line.data(), s99_delim);
    while (data != nullptr) {
      if (i == COL_TIME) {
        // in the following divide by # years per kyr (1000)
        sw_time.push_back(std::stof(std::string(data)) / 1000);
      } else if (i == COL_POWER) {
        sw_e.push_back(std::stof(std::string(data)));
      } else if (i == COL_ALL_P_FLUX) {
        sw_p.push_back(std::stof(std::string(data)));
      }
      data = strtok(nullptr, s99_delim);
      i++;
    }
  }

  time_sw_end   = sw_time[sw_time.size() - 1];
  time_sw_start = sw_time[0];
  // the following is the time interval between data points
  // (i.e. assumes regular temporal spacing)
  sw_dt = (time_sw_end - time_sw_start) / (sw_p.size() - 1);
  chprintf("wind t_s %.5e, t_e %.5e, delta T %0.5e\n", time_sw_start, time_sw_end, sw_dt);

  GPU_Error_Check(cudaMalloc((void**)&dev_sw_p, sw_p.size() * sizeof(Real)));
  GPU_Error_Check(cudaMemcpy(dev_sw_p, sw_p.data(), sw_p.size() * sizeof(Real), cudaMemcpyHostToDevice));

  GPU_Error_Check(cudaMalloc((void**)&dev_sw_e, sw_e.size() * sizeof(Real)));
  GPU_Error_Check(cudaMemcpy(dev_sw_e, sw_e.data(), sw_e.size() * sizeof(Real), cudaMemcpyHostToDevice));

  chprintf("first 40 stellar wind momentum values:\n");
  for (int i = 0; i < 40; i++) {
    chprintf("%0.5e  %5f %5f \n", sw_time.at(i), sw_e.at(i), sw_p.at(i));
  }
}

  #endif  // NO_WIND_FEEDBACK

/**
 * @brief Get the Starburst 99 stellar wind momentum flux per solar mass.
 *
 * @param t cluster age in kyr
 * @param dev_sw_p device array of log base 10 momentum flux values in dynes.
 * @param sw_dt time interval between table data points in kyr.
 * @param t_start cluster age when flux becomes non-negligible (kyr).
 * @param t_end  cluster age when stellar winds turn off (kyr).
 * @return flux (in Cholla force units) per solar mass.
 */
__device__ Real Get_Wind_Flux(Real t, Real* dev_sw_p, Real sw_dt, Real t_start, Real t_end)
{
  if (t < t_start || t >= t_end) return 0;

  int index        = (int)((t - t_start) / sw_dt);
  Real log_p_dynes = dev_sw_p[index] + (t - index * sw_dt) * (dev_sw_p[index + 1] - dev_sw_p[index]) / sw_dt;
  return pow(10, log_p_dynes) / FORCE_UNIT / S_99_TOTAL_MASS;
}

/**
 * @brief Get the Starburst 99 stellar wind emitted power per solar mass.
 *
 * @param t cluster age in kyr
 * @param dev_sw_e device array of log base 10 power (erg/s).
 * @param sw_dt time interval between table data points in kyr.
 * @param t_start cluster age when power becomes non-negligible (kyr).
 * @param t_end  cluster age when stellar winds turn off (kyr).
 * @return power (in Cholla units) per solar mass.
 */
__device__ Real Get_Wind_Power(Real t, Real* dev_sw_e, Real sw_dt, Real t_start, Real t_end)
{
  if (t < t_start || t >= t_end) return 0;

  int index  = (int)((t - t_start) / sw_dt);
  Real log_e = dev_sw_e[index] + (t - index * sw_dt) * (dev_sw_e[index + 1] - dev_sw_e[index]) / sw_dt;
  Real e     = pow(10, log_e) / (MASS_UNIT * VELOCITY_UNIT * VELOCITY_UNIT) * TIME_UNIT / S_99_TOTAL_MASS;
  return e;
}

/**
 * @brief Get the mass flux associated with stellar wind momentum flux
 *        and stellar wind power scaled per cluster mass.
 *
 * @param flux
 * @return mass flux in g/s per solar mass
 */
__device__ Real Get_Wind_Mass(Real flux, Real power)
{
  if (flux <= 0 || power <= 0) return 0;
  return flux * flux / power / 2;
}

/**
 * @brief returns SNR from starburst 99 (or default analytical rate).
 *        Time is in kyr.  Does a basic interpolation of S'99 table
 *        values.
 *
 * @param t   The cluster age.
 * @param dev_snr  device array with rate info
 * @param snr_dt  time interval between table data.  Constant value.
 * @param t_start cluster age when SNR is greater than zero.
 * @param t_end   cluster age when SNR drops to zero.
 * @return double number of SNe per kyr per solar mass
 */
__device__ Real Get_SN_Rate(Real t, Real* dev_snr, Real snr_dt, Real t_start, Real t_end)
{
  if (t < t_start || t >= t_end) return 0;
  if (dev_snr == nullptr) return feedback::DEFAULT_SNR;

  int index = (int)((t - t_start) / snr_dt);
  return dev_snr[index] + (t - index * snr_dt) * (dev_snr[index + 1] - dev_snr[index]) / snr_dt;
}

/**
 * @brief Get an actual number of SNe given the expected number.
 * Both the simulation step number and cluster ID is used to
 * set the state of the random number generator in a unique and
 * deterministic way.
 *
 * @param ave_num_sn expected number of SN, based on cluster
 * age, mass and time step.
 * @param n_step sim step number
 * @param cluster_id
 * @return number of supernovae
 */
inline __device__ int Get_Number_Of_SNe_In_Cluster(Real ave_num_sn, int n_step, part_int_t cluster_id)
{
  feedback_prng_t state;
  curand_init(FEEDBACK_SEED, 0, 0, &state);
  unsigned long long skip = n_step * 10000 + cluster_id;
  skipahead(skip, &state);  // provided by curand
  return (int)curand_poisson(&state, ave_num_sn);
}

__device__ Real Apply_Resolved_SN(Real pos_x, Real pos_y, Real pos_z, Real xMin, Real yMin, Real zMin, Real dx, Real dy,
                                  Real dz, int nx_g, int ny_g, int n_ghost, int n_cells, Real gamma,
                                  Real* conserved_device, short time_direction, Real feedback_density,
                                  Real feedback_energy)
{
  // For 2x2x2, a particle between 0-0.5 injects onto cell - 1
  int indx_x = (int)floor((pos_x - xMin - 0.5 * dx) / dx);
  int indx_y = (int)floor((pos_y - yMin - 0.5 * dy) / dy);
  int indx_z = (int)floor((pos_z - zMin - 0.5 * dz) / dz);

  Real cell_center_x = xMin + indx_x * dx + 0.5 * dx;
  Real cell_center_y = yMin + indx_y * dy + 0.5 * dy;
  Real cell_center_z = zMin + indx_z * dz + 0.5 * dz;

  Real delta_x = 1 - (pos_x - cell_center_x) / dx;
  Real delta_y = 1 - (pos_y - cell_center_y) / dy;
  Real delta_z = 1 - (pos_z - cell_center_z) / dz;

  Real* density    = conserved_device;
  Real* momentum_x = &conserved_device[n_cells * grid_enum::momentum_x];
  Real* momentum_y = &conserved_device[n_cells * grid_enum::momentum_y];
  Real* momentum_z = &conserved_device[n_cells * grid_enum::momentum_z];
  Real* energy     = &conserved_device[n_cells * grid_enum::Energy];
  Real* gasEnergy  = &conserved_device[n_cells * grid_enum::GasEnergy];

  Real local_dti = 0;

  for (int i = 0; i < 2; i++) {
    for (int j = 0; j < 2; j++) {
      for (int k = 0; k < 2; k++) {
        int indx    = (indx_x + i + n_ghost) + (indx_y + j + n_ghost) * nx_g + (indx_z + k + n_ghost) * nx_g * ny_g;
        Real x_frac = i * (1 - delta_x) + (1 - i) * delta_x;
        Real y_frac = j * (1 - delta_y) + (1 - j) * delta_y;
        Real z_frac = k * (1 - delta_z) + (1 - k) * delta_z;

        atomicAdd(&density[indx], x_frac * y_frac * z_frac * feedback_density);
        atomicAdd(&gasEnergy[indx], x_frac * y_frac * z_frac * feedback_energy);
        atomicAdd(&energy[indx], x_frac * y_frac * z_frac * feedback_energy);

        if (time_direction > 0) {
          Real cell_dti = Calc_Timestep(gamma, density, momentum_x, momentum_y, momentum_z, energy, indx, dx, dy, dz);

          local_dti = fmax(local_dti, cell_dti);
        }
      }  // k loop
    }    // j loop
  }      // i loop

  return local_dti;
}

__device__ Real Apply_Unresolved_SN(Real pos_x, Real pos_y, Real pos_z, Real xMin, Real yMin, Real zMin, Real dx,
                                    Real dy, Real dz, int nx_g, int ny_g, int n_ghost, int n_cells, Real gamma,
                                    Real* conserved_device, short time_direction, Real feedback_density,
                                    Real feedback_momentum, Real feedback_energy, int indx_x, int indx_y, int indx_z)
{
  Real delta_x = (pos_x - xMin - indx_x * dx) / dx;
  Real delta_y = (pos_y - yMin - indx_y * dy) / dy;
  Real delta_z = (pos_z - zMin - indx_z * dz) / dz;

  Real local_dti = 0;

  Real* density    = conserved_device;
  Real* momentum_x = &conserved_device[n_cells * grid_enum::momentum_x];
  Real* momentum_y = &conserved_device[n_cells * grid_enum::momentum_y];
  Real* momentum_z = &conserved_device[n_cells * grid_enum::momentum_z];
  Real* energy     = &conserved_device[n_cells * grid_enum::Energy];
  Real* gas_energy = &conserved_device[n_cells * grid_enum::GasEnergy];

  Real x_frac, y_frac, z_frac;
  Real mag = 0;
  for (int i = -1; i < 2; i++) {
    for (int j = -1; j < 2; j++) {
      for (int k = -1; k < 2; k++) {
        x_frac = D_Frac(i, delta_x) * Frac(j, delta_y) * Frac(k, delta_z);
        y_frac = Frac(i, delta_x) * D_Frac(j, delta_y) * Frac(k, delta_z);
        z_frac = Frac(i, delta_x) * Frac(j, delta_y) * D_Frac(k, delta_z);

        mag += sqrt(x_frac * x_frac + y_frac * y_frac + z_frac * z_frac);
      }
    }
  }

  for (int i = -1; i < 2; i++) {
    for (int j = -1; j < 2; j++) {
      for (int k = -1; k < 2; k++) {
        // index in array of conserved quantities
        int indx = (indx_x + i + n_ghost) + (indx_y + j + n_ghost) * nx_g + (indx_z + k + n_ghost) * nx_g * ny_g;

        x_frac = D_Frac(i, delta_x) * Frac(j, delta_y) * Frac(k, delta_z);
        y_frac = Frac(i, delta_x) * D_Frac(j, delta_y) * Frac(k, delta_z);
        z_frac = Frac(i, delta_x) * Frac(j, delta_y) * D_Frac(k, delta_z);

        Real px = x_frac * feedback_momentum;
        Real py = y_frac * feedback_momentum;
        Real pz = z_frac * feedback_momentum;
        Real d  = sqrt(x_frac * x_frac + y_frac * y_frac + z_frac * z_frac) / mag * feedback_density;
        Real e  = sqrt(x_frac * x_frac + y_frac * y_frac + z_frac * z_frac) / mag * feedback_energy;

        atomicAdd(&momentum_x[indx], px);
        atomicAdd(&momentum_y[indx], py);
        atomicAdd(&momentum_z[indx], pz);
        atomicAdd(&energy[indx], e);
        atomicAdd(&density[indx], d);

        gas_energy[indx] = energy[indx] - (momentum_x[indx] * momentum_x[indx] + momentum_y[indx] * momentum_y[indx] +
                                           momentum_z[indx] * momentum_z[indx]) /
                                              (2 * density[indx]);

        if (time_direction > 0) {
          Real cell_dti = Calc_Timestep(gamma, density, momentum_x, momentum_y, momentum_z, energy, indx, dx, dy, dz);
          local_dti     = fmax(local_dti, cell_dti);
        }
      }  // k loop
    }    // j loop
  }      // i loop

  return local_dti;
}

__device__ Real Apply_Wind(Real pos_x, Real pos_y, Real pos_z, Real xMin, Real yMin, Real zMin, Real dx, Real dy,
                           Real dz, int nx_g, int ny_g, int n_ghost, int n_cells, Real gamma, Real* conserved_device,
                           short time_direction, Real feedback_density, Real feedback_momentum, Real feedback_energy,
                           int n_step, part_int_t id, int loop, int indx_x, int indx_y, int indx_z)
{
  Real delta_x = (pos_x - xMin - indx_x * dx) / dx;
  Real delta_y = (pos_y - yMin - indx_y * dy) / dy;
  Real delta_z = (pos_z - zMin - indx_z * dz) / dz;

  Real local_dti = 0;
  Real f_energy, x_frac, y_frac, z_frac, f_dens;

  Real* density    = conserved_device;
  Real* momentum_x = &conserved_device[n_cells * grid_enum::momentum_x];
  Real* momentum_y = &conserved_device[n_cells * grid_enum::momentum_y];
  Real* momentum_z = &conserved_device[n_cells * grid_enum::momentum_z];
  Real* energy     = &conserved_device[n_cells * grid_enum::Energy];
  Real* gas_energy = &conserved_device[n_cells * grid_enum::GasEnergy];

  // loop over the 27 cells to add up all the allocated feedback
  // momentum magnitudes.  For each cell allocate density and
  // energy based on the ratio of allocated momentum to this overall sum.
  Real mag = 0;
  for (int i = -1; i < 2; i++) {
    for (int j = -1; j < 2; j++) {
      for (int k = -1; k < 2; k++) {
        x_frac = D_Frac(i, delta_x) * Frac(j, delta_y) * Frac(k, delta_z);
        y_frac = Frac(i, delta_x) * D_Frac(j, delta_y) * Frac(k, delta_z);
        z_frac = Frac(i, delta_x) * Frac(j, delta_y) * D_Frac(k, delta_z);

        mag += sqrt(x_frac * x_frac + y_frac * y_frac + z_frac * z_frac);
      }
    }
  }

  for (int i = -1; i < 2; i++) {
    for (int j = -1; j < 2; j++) {
      for (int k = -1; k < 2; k++) {
        // index in array of conserved quantities
        int indx = (indx_x + i + n_ghost) + (indx_y + j + n_ghost) * nx_g + (indx_z + k + n_ghost) * nx_g * ny_g;

        x_frac = D_Frac(i, delta_x) * Frac(j, delta_y) * Frac(k, delta_z);
        y_frac = Frac(i, delta_x) * D_Frac(j, delta_y) * Frac(k, delta_z);
        z_frac = Frac(i, delta_x) * Frac(j, delta_y) * D_Frac(k, delta_z);

        Real px  = x_frac * feedback_momentum;
        Real py  = y_frac * feedback_momentum;
        Real pz  = z_frac * feedback_momentum;
        f_dens   = sqrt(x_frac * x_frac + y_frac * y_frac + z_frac * z_frac) / mag * feedback_density;
        f_energy = sqrt(x_frac * x_frac + y_frac * y_frac + z_frac * z_frac) / mag * feedback_energy;

        atomicAdd(&density[indx], f_dens);
        atomicAdd(&momentum_x[indx], px);
        atomicAdd(&momentum_y[indx], py);
        atomicAdd(&momentum_z[indx], pz);
        atomicAdd(&energy[indx], f_energy);

        gas_energy[indx] = energy[indx] - (momentum_x[indx] * momentum_x[indx] + momentum_y[indx] * momentum_y[indx] +
                                           momentum_z[indx] * momentum_z[indx]) /
                                              (2 * density[indx]);

        /*
        energy[indx] = ( momentum_x[indx] * momentum_x[indx] +
                         momentum_y[indx] * momentum_y[indx] +
                         momentum_z[indx] * momentum_z[indx] ) /
                       2 / density[indx] + gasEnergy[indx];
        */
        if (time_direction > 0) {
          Real cell_dti = Calc_Timestep(gamma, density, momentum_x, momentum_y, momentum_z, energy, indx, dx, dy, dz);
          local_dti     = fmax(local_dti, cell_dti);
        }

      }  // k loop
    }    // j loop
  }      // i loop

  return local_dti;
}

__device__ void SN_Feedback(Real pos_x, Real pos_y, Real pos_z, Real age, Real* mass_dev, part_int_t* id_dev, Real xMin,
                            Real yMin, Real zMin, Real xMax, Real yMax, Real zMax, Real dx, Real dy, Real dz, int nx_g,
                            int ny_g, int nz_g, int n_ghost, int n_step, Real t, Real dt, Real* dti, Real* dev_snr,
                            Real snr_dt, Real time_sn_start, Real time_sn_end, Real* prev_dens, short time_direction,
                            Real* s_info, Real* conserved_dev, Real gamma, int loop, int indx_x, int indx_y, int indx_z)
{
  int tid  = threadIdx.x;
  int gtid = blockIdx.x * blockDim.x + tid;

  Real dV = dx * dy * dz;
  Real feedback_density, feedback_momentum, feedback_energy;
  Real local_dti = 0.0;
  int n_cells    = nx_g * ny_g * nz_g;

  Real average_num_sn = Get_SN_Rate(age, dev_snr, snr_dt, time_sn_start, time_sn_end) * mass_dev[gtid] * dt;
  int N               = Get_Number_Of_SNe_In_Cluster(average_num_sn, n_step, id_dev[gtid]) * time_direction;
  /*
  if (gtid == 0) {
    kernel_printf("SNUMBER n_step: %d, id: %lld, N: %d\n", n_step, id_dev[gtid], N);
  }
  */

  // no sense doing anything if there was no SN
  if (N != 0) {
    Real n_0;
    if (time_direction == -1) {
      n_0 = prev_dens[gtid];
    } else {
      Real* density             = conserved_dev;
      n_0                       = Get_Average_Number_Density_CGS(density, indx_x, indx_y, indx_z, nx_g, ny_g, n_ghost);
      prev_dens[gtid]           = n_0;
      s_info[FEED_INFO_N * tid] = 1. * N;
    }

    feedback_energy  = N * feedback::ENERGY_PER_SN / dV;
    feedback_density = N * feedback::MASS_PER_SN / dV;

    Real shell_radius = feedback::R_SH * pow(n_0, -0.46) * pow(fabsf(N), 0.29);
  #ifdef ONLY_RESOLVED
    bool is_resolved = true;
  #else
    bool is_resolved = 3 * max(dx, max(dy, dz)) <= shell_radius;
  #endif

    if (is_resolved) {
      // inject energy and density
      if (time_direction > 0) {
        s_info[FEED_INFO_N * tid + i_RES]    = 1. * N;
        s_info[FEED_INFO_N * tid + i_ENERGY] = feedback_energy * dV;
      }
      local_dti = Apply_Resolved_SN(pos_x, pos_y, pos_z, xMin, yMin, zMin, dx, dy, dz, nx_g, ny_g, n_ghost, n_cells,
                                    gamma, conserved_dev, time_direction, feedback_density, feedback_energy);
    } else {
      // inject momentum and density
      feedback_momentum =
          time_direction * feedback::FINAL_MOMENTUM * pow(n_0, -0.17) * pow(fabsf(N), 0.93) / dV / sqrt(3.0);
      if (time_direction > 0) {
        s_info[FEED_INFO_N * tid + i_UNRES]        = 1. * N;
        s_info[FEED_INFO_N * tid + i_MOMENTUM]     = feedback_momentum * dV * sqrt(3.0);
        s_info[FEED_INFO_N * tid + i_UNRES_ENERGY] = feedback_energy * dV;
      }
      local_dti = Apply_Unresolved_SN(pos_x, pos_y, pos_z, xMin, yMin, zMin, dx, dy, dz, nx_g, ny_g, n_ghost, n_cells,
                                      gamma, conserved_dev, time_direction, feedback_density, feedback_momentum,
                                      feedback_energy, indx_x, indx_y, indx_z);
    }
  }

  if (time_direction > 0) atomicMax(dti, local_dti);
}

__device__ void Wind_Feedback(Real pos_x, Real pos_y, Real pos_z, Real age, Real* mass_dev, part_int_t* id_dev,
                              Real xMin, Real yMin, Real zMin, Real xMax, Real yMax, Real zMax, Real dx, Real dy,
                              Real dz, int nx_g, int ny_g, int nz_g, int n_ghost, int n_step, Real t, Real dt,
                              Real* dti, Real* dev_sw_p, Real* dev_sw_e, Real sw_dt, Real time_sw_start,
                              Real time_sw_end, short time_direction, Real* s_info, Real* conserved_dev, Real gamma,
                              int loop, int indx_x, int indx_y, int indx_z)
{
  int tid  = threadIdx.x;
  int gtid = blockIdx.x * blockDim.x + tid;

  Real dV = dx * dy * dz;
  Real feedback_density, feedback_momentum, feedback_energy;
  Real local_dti = 0.0;
  int n_cells    = nx_g * ny_g * nz_g;

  if (age < 0 || age > time_sw_end) return;
  feedback_momentum = Get_Wind_Flux(age, dev_sw_p, sw_dt, time_sw_start, time_sw_end);
  // no sense in proceeding if there is no feedback.
  if (feedback_momentum == 0) return;
  feedback_energy  = Get_Wind_Power(age, dev_sw_e, sw_dt, time_sw_start, time_sw_end);
  feedback_density = Get_Wind_Mass(feedback_momentum, feedback_energy);

  // feedback_momentum now becomes momentum component along one direction.
  feedback_momentum *= mass_dev[gtid] * dt / dV / sqrt(3.0) * time_direction;
  feedback_density *= mass_dev[gtid] * dt / dV * time_direction;
  feedback_energy *= mass_dev[gtid] * dt / dV * time_direction;

  /* TODO refactor into separate kernel call
  if (time_direction > 0) {
    mass_dev[gtid]   -= feedback_density * dV;
  }*/

  if (time_direction > 0) {
    // we log net momentum, not momentum density, and magnitude (not the
    // component along a direction)
    s_info[FEED_INFO_N * tid + i_WIND_MOMENTUM] = feedback_momentum * dV * sqrt(3.0);
    s_info[FEED_INFO_N * tid + i_WIND_ENERGY]   = feedback_energy * dV;
  }

  local_dti = Apply_Wind(pos_x, pos_y, pos_z, xMin, yMin, zMin, dx, dy, dz, nx_g, ny_g, n_ghost, n_cells, gamma,
                         conserved_dev, time_direction, feedback_density, feedback_momentum, feedback_energy, n_step,
                         id_dev[gtid], loop, indx_x, indx_y, indx_z);

  if (time_direction > 0) atomicMax(dti, local_dti);
}

__device__ void Cluster_Feedback_Helper(part_int_t n_local, Real* pos_x_dev, Real* pos_y_dev, Real* pos_z_dev,
                                        Real* age_dev, Real* mass_dev, part_int_t* id_dev, Real xMin, Real yMin,
                                        Real zMin, Real xMax, Real yMax, Real zMax, Real dx, Real dy, Real dz, int nx_g,
                                        int ny_g, int nz_g, int n_ghost, int n_step, Real t, Real dt, Real* dti,
                                        Real* dev_snr, Real snr_dt, Real time_sn_start, Real time_sn_end,
                                        Real* prev_dens, Real* dev_sw_p, Real* dev_sw_e, Real sw_dt, Real time_sw_start,
                                        Real time_sw_end, short time_direction, Real* s_info, Real* conserved_dev,
                                        Real gamma, int loop)
{
  int tid  = threadIdx.x;
  int gtid = blockIdx.x * blockDim.x + tid;
  // Bounds check on particle arrays
  if (gtid >= n_local) return;

  Real pos_x    = pos_x_dev[gtid];
  Real pos_y    = pos_y_dev[gtid];
  Real pos_z    = pos_z_dev[gtid];
  bool in_local = (pos_x >= xMin && pos_x < xMax) && (pos_y >= yMin && pos_y < yMax) && (pos_z >= zMin && pos_z < zMax);
  // Particle is outside bounds, exit
  if (!in_local) return;

  int indx_x  = (int)floor((pos_x - xMin) / dx);
  int indx_y  = (int)floor((pos_y - yMin) / dy);
  int indx_z  = (int)floor((pos_z - zMin) / dz);
  bool ignore = indx_x < 0 || indx_y < 0 || indx_z < 0 || indx_x >= nx_g - 2 * n_ghost ||
                indx_y >= ny_g - 2 * n_ghost || indx_z >= nz_g - 2 * n_ghost;
  // Ignore this particle, exit
  if (ignore) return;

  // bool is_alone = Particle_Is_Alone(pos_x_dev, pos_y_dev, pos_z_dev, n_local, gtid, 6*dx);
  // if (is_alone) kernel_printf(" particle not alone: step %d, id %ld\n", n_step, id_dev[gtid]);
  // if (!is_alone) return;

  // note age_dev is actually the time of birth
  Real age = t - age_dev[gtid];

  bool is_sn_feedback = false;
  bool is_wd_feedback = false;
  #ifndef NO_SN_FEEDBACK
  is_sn_feedback = true;
  #endif
  #ifndef NO_WIND_FEEDBACK
  is_wd_feedback = true;
  #endif

  // when applying different types of feedback, undoing the step requires
  // reverising the order
  if (time_direction > 0) {
    if (is_sn_feedback) {
      SN_Feedback(pos_x, pos_y, pos_z, age, mass_dev, id_dev, xMin, yMin, zMin, xMax, yMax, zMax, dx, dy, dz, nx_g,
                  ny_g, nz_g, n_ghost, n_step, t, dt, dti, dev_snr, snr_dt, time_sn_start, time_sn_end, prev_dens,
                  time_direction, s_info, conserved_dev, gamma, loop, indx_x, indx_y, indx_z);
    }
    if (is_wd_feedback) {
      Wind_Feedback(pos_x, pos_y, pos_z, age, mass_dev, id_dev, xMin, yMin, zMin, xMax, yMax, zMax, dx, dy, dz, nx_g,
                    ny_g, nz_g, n_ghost, n_step, t, dt, dti, dev_sw_p, dev_sw_e, sw_dt, time_sw_start, time_sw_end,
                    time_direction, s_info, conserved_dev, gamma, loop, indx_x, indx_y, indx_z);
    }
  } else {
    if (is_wd_feedback) {
      Wind_Feedback(pos_x, pos_y, pos_z, age, mass_dev, id_dev, xMin, yMin, zMin, xMax, yMax, zMax, dx, dy, dz, nx_g,
                    ny_g, nz_g, n_ghost, n_step, t, dt, dti, dev_sw_p, dev_sw_e, sw_dt, time_sw_start, time_sw_end,
                    time_direction, s_info, conserved_dev, gamma, loop, indx_x, indx_y, indx_z);
    }
    if (is_sn_feedback) {
      SN_Feedback(pos_x, pos_y, pos_z, age, mass_dev, id_dev, xMin, yMin, zMin, xMax, yMax, zMax, dx, dy, dz, nx_g,
                  ny_g, nz_g, n_ghost, n_step, t, dt, dti, dev_snr, snr_dt, time_sn_start, time_sn_end, prev_dens,
                  time_direction, s_info, conserved_dev, gamma, loop, indx_x, indx_y, indx_z);
    }
  }

  return;
}

__global__ void Cluster_Feedback_Kernel(part_int_t n_local, part_int_t* id_dev, Real* pos_x_dev, Real* pos_y_dev,
                                        Real* pos_z_dev, Real* mass_dev, Real* age_dev, Real xMin, Real yMin, Real zMin,
                                        Real xMax, Real yMax, Real zMax, Real dx, Real dy, Real dz, int nx_g, int ny_g,
                                        int nz_g, int n_ghost, Real t, Real dt, Real* dti, Real* info, Real* density,
                                        Real gamma, Real* prev_dens, short time_direction, Real* dev_snr, Real snr_dt,
                                        Real time_sn_start, Real time_sn_end, Real* dev_sw_p, Real* dev_sw_e,
                                        Real sw_dt, Real time_sw_start, Real time_sw_end, int n_step, int loop)
{
  int tid = threadIdx.x;

  // for collecting SN feedback information
  __shared__ Real s_info[FEED_INFO_N * TPB_FEEDBACK];
  s_info[FEED_INFO_N * tid]     = 0;  // number of supernovae
  s_info[FEED_INFO_N * tid + 1] = 0;  // number of resolved events
  s_info[FEED_INFO_N * tid + 2] = 0;  // number of unresolved events
  s_info[FEED_INFO_N * tid + 3] = 0;  // resolved energy
  s_info[FEED_INFO_N * tid + 4] = 0;  // unresolved momentum
  s_info[FEED_INFO_N * tid + 5] = 0;  // unresolved KE added via momentum
  s_info[FEED_INFO_N * tid + 6] = 0;  // wind momentum
  s_info[FEED_INFO_N * tid + 7] = 0;  // wind energy added

  Cluster_Feedback_Helper(n_local, pos_x_dev, pos_y_dev, pos_z_dev, age_dev, mass_dev, id_dev, xMin, yMin, zMin, xMax,
                          yMax, zMax, dx, dy, dz, nx_g, ny_g, nz_g, n_ghost, n_step, t, dt, dti, dev_snr, snr_dt,
                          time_sn_start, time_sn_end, prev_dens, dev_sw_p, dev_sw_e, sw_dt, time_sw_start, time_sw_end,
                          time_direction, s_info, density, gamma, loop);

  __syncthreads();

  // reduce the info from all the threads in the block
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      s_info[FEED_INFO_N * tid] += s_info[FEED_INFO_N * (tid + s)];
      s_info[FEED_INFO_N * tid + 1] += s_info[FEED_INFO_N * (tid + s) + 1];
      s_info[FEED_INFO_N * tid + 2] += s_info[FEED_INFO_N * (tid + s) + 2];
      s_info[FEED_INFO_N * tid + 3] += s_info[FEED_INFO_N * (tid + s) + 3];
      s_info[FEED_INFO_N * tid + 4] += s_info[FEED_INFO_N * (tid + s) + 4];
      s_info[FEED_INFO_N * tid + 5] += s_info[FEED_INFO_N * (tid + s) + 5];
      s_info[FEED_INFO_N * tid + 6] += s_info[FEED_INFO_N * (tid + s) + 6];
      s_info[FEED_INFO_N * tid + 7] += s_info[FEED_INFO_N * (tid + s) + 7];
    }
    __syncthreads();
  }

  // atomicAdd reduces across all blocks
  if (tid == 0) {
    atomicAdd(info, s_info[0]);
    atomicAdd(info + 1, s_info[1]);
    atomicAdd(info + 2, s_info[2]);
    atomicAdd(info + 3, s_info[3]);
    atomicAdd(info + 4, s_info[4]);
    atomicAdd(info + 5, s_info[5]);
    atomicAdd(info + 6, s_info[6]);
    atomicAdd(info + 7, s_info[7]);
  }
}

__global__ void Adjust_Cluster_Mass_Kernel(part_int_t n_local, Real* pos_x_dev, Real* pos_y_dev, Real* pos_z_dev,
                                           Real* age_dev, Real* mass_dev, part_int_t* id_dev, Real xMin, Real yMin,
                                           Real zMin, Real xMax, Real yMax, Real zMax, Real dx, Real dy, Real dz,
                                           int nx_g, int ny_g, int nz_g, int n_ghost, int n_step, Real t, Real dt,
                                           Real* dev_snr, Real snr_dt, Real time_sn_start, Real time_sn_end,
                                           Real* dev_sw_p, Real* dev_sw_e, Real sw_dt, Real time_sw_start,
                                           Real time_sw_end)
{
  int tid  = threadIdx.x;
  int gtid = blockIdx.x * blockDim.x + tid;
  // Bounds check on particle arrays
  if (gtid >= n_local) return;

  Real pos_x    = pos_x_dev[gtid];
  Real pos_y    = pos_y_dev[gtid];
  Real pos_z    = pos_z_dev[gtid];
  bool in_local = (pos_x >= xMin && pos_x < xMax) && (pos_y >= yMin && pos_y < yMax) && (pos_z >= zMin && pos_z < zMax);
  // Particle is outside bounds, exit
  if (!in_local) return;

  int indx_x  = (int)floor((pos_x - xMin) / dx);
  int indx_y  = (int)floor((pos_y - yMin) / dy);
  int indx_z  = (int)floor((pos_z - zMin) / dz);
  bool ignore = indx_x < 0 || indx_y < 0 || indx_z < 0 || indx_x >= nx_g - 2 * n_ghost ||
                indx_y >= ny_g - 2 * n_ghost || indx_z >= nz_g - 2 * n_ghost;
  // Ignore this particle, exit
  if (ignore) return;

  // bool is_alone = Particle_Is_Alone(pos_x_dev, pos_y_dev, pos_z_dev, n_local, gtid, 6*dx);
  // if (is_alone) kernel_printf(" particle not alone: step %d, id %ld\n", n_step, id_dev[gtid]);
  // if (!is_alone) return;

  Real age = t - age_dev[gtid];

  #ifndef NO_SN_FEEDBACK
  Real average_num_sn = Get_SN_Rate(age, dev_snr, snr_dt, time_sn_start, time_sn_end) * mass_dev[gtid] * dt;
  int N               = Get_Number_Of_SNe_In_Cluster(average_num_sn, n_step, id_dev[gtid]);
  mass_dev[gtid] -= N * feedback::MASS_PER_SN;
  #endif

  #ifndef NO_WIND_FEEDBACK
  Real feedback_momentum  = Get_Wind_Flux(age, dev_sw_p, sw_dt, time_sw_start, time_sw_end);
  Real feedback_energy    = Get_Wind_Power(age, dev_sw_e, sw_dt, time_sw_start, time_sw_end);
  Real feedback_mass_rate = Get_Wind_Mass(feedback_momentum, feedback_energy);

  mass_dev[gtid] -= feedback_mass_rate * dt;
  #endif
}

__device__ void Set_Average_Density(int indx_x, int indx_y, int indx_z, int nx_g, int ny_g, int n_ghost, Real* density,
                                    Real ave_dens)
{
  for (int i = -1; i < 2; i++) {
    for (int j = -1; j < 2; j++) {
      for (int k = -1; k < 2; k++) {
        int indx = (indx_x + i + n_ghost) + (indx_y + j + n_ghost) * nx_g + (indx_z + k + n_ghost) * nx_g * ny_g;

        density[indx] = ave_dens;
      }
    }
  }
}

__global__ void Set_Ave_Density_Kernel(part_int_t n_local, Real* pos_x_dev, Real* pos_y_dev, Real* pos_z_dev,
                                       Real* mass_dev, Real* age_dev, part_int_t* id_dev, Real xMin, Real yMin,
                                       Real zMin, Real xMax, Real yMax, Real zMax, Real dx, Real dy, Real dz, int nx_g,
                                       int ny_g, int nz_g, int n_ghost, Real t, Real dt, Real* density, Real* dev_snr,
                                       Real snr_dt, Real time_sn_start, Real time_sn_end, Real time_sw_start,
                                       Real time_sw_end, int n_step)
{
  int tid  = threadIdx.x;
  int gtid = blockIdx.x * blockDim.x + tid;
  // Bounds check on particle arrays
  if (gtid >= n_local) return;

  Real pos_x    = pos_x_dev[gtid];
  Real pos_y    = pos_y_dev[gtid];
  Real pos_z    = pos_z_dev[gtid];
  bool in_local = (pos_x >= xMin && pos_x < xMax) && (pos_y >= yMin && pos_y < yMax) && (pos_z >= zMin && pos_z < zMax);
  // Particle is outside bounds, exit
  if (!in_local) return;

  int indx_x  = (int)floor((pos_x - xMin) / dx);
  int indx_y  = (int)floor((pos_y - yMin) / dy);
  int indx_z  = (int)floor((pos_z - zMin) / dz);
  bool ignore = indx_x < 0 || indx_y < 0 || indx_z < 0 || indx_x >= nx_g - 2 * n_ghost ||
                indx_y >= ny_g - 2 * n_ghost || indx_z >= nz_g - 2 * n_ghost;
  // Ignore this particle, exit
  if (ignore) return;

  // bool is_alone = Particle_Is_Alone(pos_x_dev, pos_y_dev, pos_z_dev, n_local, gtid, 6*dx);
  // if (is_alone) kernel_printf(" particle not alone: step %d, id %ld\n", n_step, id_dev[gtid]);
  // if (!is_alone) return;

  bool is_sn_feedback   = false;
  bool is_wind_feedback = false;
  #ifndef NO_SN_FEEDBACK
  is_sn_feedback = true;
  #endif
  #ifndef NO_WIND_FEEDBACK
  is_wind_feedback = true;
  #endif

  Real ave_dens;
  Real age = t - age_dev[gtid];
  if (is_wind_feedback) {
    if (time_sw_start <= age && age <= time_sw_end) {
      ave_dens = Get_Average_Density(density, indx_x, indx_y, indx_z, nx_g, ny_g, n_ghost);
      Set_Average_Density(indx_x, indx_y, indx_z, nx_g, ny_g, n_ghost, density, ave_dens);
      // since we've set the average density, no need to keep
      // checking whether we should do so.
      return;
    }
  }
  if (is_sn_feedback) {
    if (time_sn_start <= age && age <= time_sn_end) {
      Real average_num_sn = Get_SN_Rate(age, dev_snr, snr_dt, time_sn_start, time_sn_end) * mass_dev[gtid] * dt;
      int N               = Get_Number_Of_SNe_In_Cluster(average_num_sn, n_step, id_dev[gtid]);
      /*
      if (gtid == 0) {
        kernel_printf("AVEDENS n_step: %d, id: %lld, N: %d\n", n_step, id_dev[gtid], N);
      }*/
      Real n_0          = Get_Average_Number_Density_CGS(density, indx_x, indx_y, indx_z, nx_g, ny_g, n_ghost);
      Real shell_radius = feedback::R_SH * pow(n_0, -0.46) * pow(N, 0.29);
  #ifdef ONLY_RESOLVED
      bool is_resolved = true;
  #else
      bool is_resolved = 3 * max(dx, max(dy, dz)) <= shell_radius;
  #endif

      // resolved SN feedback does not average densities.
      if (!is_resolved && N > 0) {
        ave_dens = n_0 * MU * MP / DENSITY_UNIT;
        Set_Average_Density(indx_x, indx_y, indx_z, nx_g, ny_g, n_ghost, density, ave_dens);
      }
    }
  }
}

/**
 * @brief Stellar feedback function (SNe and stellar winds)
 *
 * @param G
 * @param analysis
 * @return Real
 */
Real feedback::Cluster_Feedback(Grid3D& G, FeedbackAnalysis& analysis)
{
  #ifdef CPU_TIME
  G.Timer.Feedback.Start();
  #endif

  if (G.H.dt == 0) return 0.0;

  Real h_dti = 0.0;
  int time_direction, ngrid;
  Real h_info[FEED_INFO_N] = {0, 0, 0, 0, 0, 0, 0, 0};
  Real info[FEED_INFO_N];
  Real *d_dti, *d_info;
  // require d_prev_dens in case we have to undo feedback if the time
  // step is too large.
  Real* d_prev_dens;

  // only apply feedback if we have clusters
  if (G.Particles.n_local > 0) {
    GPU_Error_Check(cudaMalloc(&d_dti, sizeof(Real)));
    GPU_Error_Check(cudaMemcpy(d_dti, &h_dti, sizeof(Real), cudaMemcpyHostToDevice));
    GPU_Error_Check(cudaMalloc(&d_prev_dens, G.Particles.n_local * sizeof(Real)));
    GPU_Error_Check(cudaMemset(d_prev_dens, 0, G.Particles.n_local * sizeof(Real)));

    ngrid = (G.Particles.n_local - 1) / TPB_FEEDBACK + 1;
    GPU_Error_Check(cudaMalloc((void**)&d_info, FEED_INFO_N * sizeof(Real)));

    // before applying feedback, set gas density around clusters to the
    // average value from the 27 neighboring cells.  We don't want to
    // do this during application of feedback since "undoing it" in the
    // event that the time step is too large becomes difficult.
    hipLaunchKernelGGL(Set_Ave_Density_Kernel, ngrid, TPB_FEEDBACK, 0, 0, G.Particles.n_local, G.Particles.pos_x_dev,
                       G.Particles.pos_y_dev, G.Particles.pos_z_dev, G.Particles.mass_dev, G.Particles.age_dev,
                       G.Particles.partIDs_dev, G.H.xblocal, G.H.yblocal, G.H.zblocal, G.H.xblocal_max, G.H.yblocal_max,
                       G.H.zblocal_max, G.H.dx, G.H.dy, G.H.dz, G.H.nx, G.H.ny, G.H.nz, G.H.n_ghost, G.H.t, G.H.dt,
                       G.C.d_density, dev_snr, snr_dt, time_sn_start, time_sn_end, time_sw_start, time_sw_end,
                       G.H.n_step);
  }

  int loop_counter = 0;

  do {
    time_direction = 1;
    loop_counter++;

    if (G.Particles.n_local > 0) {
      // always reset d_info to 0 since otherwise do/while looping could add
      // values that should have been reverted.
      cudaMemset(d_info, 0, FEED_INFO_N * sizeof(Real));
      cudaMemset(d_dti, 0, sizeof(Real));
      hipLaunchKernelGGL(Cluster_Feedback_Kernel, ngrid, TPB_FEEDBACK, 0, 0, G.Particles.n_local,
                         G.Particles.partIDs_dev, G.Particles.pos_x_dev, G.Particles.pos_y_dev, G.Particles.pos_z_dev,
                         G.Particles.mass_dev, G.Particles.age_dev, G.H.xblocal, G.H.yblocal, G.H.zblocal,
                         G.H.xblocal_max, G.H.yblocal_max, G.H.zblocal_max, G.H.dx, G.H.dy, G.H.dz, G.H.nx, G.H.ny,
                         G.H.nz, G.H.n_ghost, G.H.t, G.H.dt, d_dti, d_info, G.C.d_density, gama, d_prev_dens,
                         time_direction, dev_snr, snr_dt, time_sn_start, time_sn_end, dev_sw_p, dev_sw_e, sw_dt,
                         time_sw_start, time_sw_end, G.H.n_step, loop_counter);

      GPU_Error_Check(cudaMemcpy(&h_dti, d_dti, sizeof(Real), cudaMemcpyDeviceToHost));
    }

  #ifdef MPI_CHOLLA
    h_dti = ReduceRealMax(h_dti);
    MPI_Barrier(world);
  #endif  // MPI_CHOLLA
    if (h_dti != 0) {
      chprintf("+++++++  feed dt = %.12e, H.dt = %.12e\n", C_cfl / h_dti, G.H.dt);
    }

    if (h_dti != 0 && (C_cfl / h_dti < G.H.dt)) {
      // timestep too big: need to undo the last operation
      time_direction = -1;
      if (G.Particles.n_local > 0) {
        hipLaunchKernelGGL(Cluster_Feedback_Kernel, ngrid, TPB_FEEDBACK, 0, 0, G.Particles.n_local,
                           G.Particles.partIDs_dev, G.Particles.pos_x_dev, G.Particles.pos_y_dev, G.Particles.pos_z_dev,
                           G.Particles.mass_dev, G.Particles.age_dev, G.H.xblocal, G.H.yblocal, G.H.zblocal,
                           G.H.xblocal_max, G.H.yblocal_max, G.H.zblocal_max, G.H.dx, G.H.dy, G.H.dz, G.H.nx, G.H.ny,
                           G.H.nz, G.H.n_ghost, G.H.t, G.H.dt, d_dti, d_info, G.C.d_density, gama, d_prev_dens,
                           time_direction, dev_snr, snr_dt, time_sn_start, time_sn_end, dev_sw_p, dev_sw_e, sw_dt,
                           time_sw_start, time_sw_end, G.H.n_step, loop_counter);

        GPU_Error_Check(cudaDeviceSynchronize());
      }

      G.H.dt = C_cfl / h_dti;
      if (loop_counter > 2) {  // avoid excessive looping
        G.H.dt = 0.9 * C_cfl / h_dti;
      }
    }
  } while (time_direction == -1);

  // TODO reduce cluster mass
  if (G.Particles.n_local > 0) {
    hipLaunchKernelGGL(Adjust_Cluster_Mass_Kernel, ngrid, TPB_FEEDBACK, 0, 0, G.Particles.n_local,
                       G.Particles.pos_x_dev, G.Particles.pos_y_dev, G.Particles.pos_z_dev, G.Particles.age_dev,
                       G.Particles.mass_dev, G.Particles.partIDs_dev, G.H.xblocal, G.H.yblocal, G.H.zblocal,
                       G.H.xblocal_max, G.H.yblocal_max, G.H.zblocal_max, G.H.dx, G.H.dy, G.H.dz, G.H.nx, G.H.ny,
                       G.H.nz, G.H.n_ghost, G.H.n_step, G.H.t, G.H.dt, dev_snr, snr_dt, time_sn_start, time_sn_end,
                       dev_sw_p, dev_sw_e, sw_dt, time_sw_start, time_sw_end);
  }
  /*
  part_int_t n_local, Real* pos_x_dev,
  Real* pos_y_dev, Real* pos_z_dev, Real* age_dev, Real* mass_dev,
  part_int_t* id_dev, Real xMin, Real yMin, Real zMin, Real xMax, Real yMax,
  Real zMax, Real dx, Real dy, Real dz, int nx_g, int ny_g, int nz_g,
  int n_ghost, int n_step, Real t, Real dt, Real* dev_snr,
  Real snr_dt, Real time_sn_start, Real time_sn_end,
  Real* dev_sw_p, Real* dev_sw_e, Real sw_dt, Real time_sw_start,
  Real time_sw_end*/

  chprintf("*******  looped %d time(s)\n", loop_counter);

  if (G.Particles.n_local > 0) {
    GPU_Error_Check(cudaMemcpy(&h_info, d_info, FEED_INFO_N * sizeof(Real), cudaMemcpyDeviceToHost));
    GPU_Error_Check(cudaFree(d_dti));
    GPU_Error_Check(cudaFree(d_info));
    GPU_Error_Check(cudaFree(d_prev_dens));
  }

  #ifdef MPI_CHOLLA
  MPI_Reduce(&h_info, &info, FEED_INFO_N, MPI_CHREAL, MPI_SUM, root, world);
  #else
  info = h_info;
  #endif

  #ifdef MPI_CHOLLA  // only do stats gathering on root rank
  if (procID == 0) {
  #endif

    analysis.countSN += (long)info[feedback::SN];
    analysis.countResolved += (long)info[feedback::RESOLVED];
    analysis.countUnresolved += (long)info[feedback::NOT_RESOLVED];
    analysis.totalEnergy += info[feedback::ENERGY];
    analysis.totalMomentum += info[feedback::MOMENTUM];
    analysis.totalUnresEnergy += info[feedback::UNRES_ENERGY];
    analysis.totalWindMomentum += info[i_WIND_MOMENTUM];
    analysis.totalWindEnergy += info[i_WIND_ENERGY];

    chprintf("iteration %d, t %.4e, dt %.4e", G.H.n_step, G.H.t, G.H.dt);

  #ifndef NO_SN_FEEDBACK
    Real global_resolved_ratio = 0.0;
    if (analysis.countResolved > 0 || analysis.countUnresolved > 0) {
      global_resolved_ratio = analysis.countResolved / double(analysis.countResolved + analysis.countUnresolved);
    }
    chprintf(": number of SN: %d,(R: %d, UR: %d)\n", (int)info[feedback::SN], (long)info[feedback::RESOLVED],
             (long)info[feedback::NOT_RESOLVED]);
    chprintf("    cummulative: #SN: %d, ratio of resolved (R: %d, UR: %d) = %.3e\n", (long)analysis.countSN,
             (long)analysis.countResolved, (long)analysis.countUnresolved, global_resolved_ratio);
    chprintf("    sn  r energy  : %.5e erg, cumulative: %.5e erg\n", info[feedback::ENERGY] * FORCE_UNIT * LENGTH_UNIT,
             analysis.totalEnergy * FORCE_UNIT * LENGTH_UNIT);
    chprintf("    sn ur energy  : %.5e erg, cumulative: %.5e erg\n",
             info[feedback::UNRES_ENERGY] * FORCE_UNIT * LENGTH_UNIT,
             analysis.totalUnresEnergy * FORCE_UNIT * LENGTH_UNIT);
    chprintf("    sn momentum  : %.5e SM km/s, cumulative: %.5e SM km/s\n",
             info[feedback::MOMENTUM] * VELOCITY_UNIT / 1e5, analysis.totalMomentum * VELOCITY_UNIT / 1e5);
  #endif  // NO_SN_FEEDBACK

  #ifndef NO_WIND_FEEDBACK
    chprintf("    wind momentum: %.5e S.M. km/s,  cumulative: %.5e S.M. km/s\n",
             info[i_WIND_MOMENTUM] * VELOCITY_UNIT / 1e5, analysis.totalWindMomentum * VELOCITY_UNIT / 1e5);
    chprintf("    wind energy  : %.5e erg,  cumulative: %.5e erg\n", info[i_WIND_ENERGY] * FORCE_UNIT * LENGTH_UNIT,
             analysis.totalWindEnergy * FORCE_UNIT * LENGTH_UNIT);
  #endif  // NO_WIND_FEEDBACK

  #ifdef MPI_CHOLLA
  }  //   end if procID == 0
  #endif

  #ifdef CPU_TIME
  G.Timer.Feedback.End();
  #endif

  return h_dti;
}

#endif  // FEEDBACK & PARTICLES_GPU & PARTICLE_IDS & PARTICLE_AGE
