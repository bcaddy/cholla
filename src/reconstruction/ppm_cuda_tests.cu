/*!
 * \file ppm_cuda_tests.cu
 * \brief Tests for the contents of ppm_cuda.h and ppm_cuda.cu
 *
 */

// STL Includes
#include <algorithm>
#include <random>
#include <string>
#include <unordered_map>
#include <vector>

// External Includes
#include <gtest/gtest.h>  // Include GoogleTest and related libraries/headers

// Local Includes
#include "../global/global.h"
#include "../io/io.h"
#include "../reconstruction/ppm_cuda.h"
#include "../utils/DeviceVector.h"
#include "../utils/hydro_utilities.h"
#include "../utils/testing_utilities.h"

TEST(tALLPpmReconstructor, CorrectInputExpectCorrectOutput)
{
#ifdef DE
  /// This test doesn't support Dual Energy. It wouldn't be that hard to add support for DE but the DE parts of the
  /// reconstructor (loading and PPM_Single_Variable) are well tested elsewhere so there's no need to add the extra
  /// complexity here.
  GTEST_SKIP();
#endif  // DE

  // Set up PRNG to use
  std::mt19937_64 prng(42);
  std::uniform_real_distribution<double> doubleRand(0.1, 5);

  // Mock up needed information
  size_t const nx    = 6;
  size_t const ny    = 6;
  size_t const nz    = 6;
  double const gamma = 5.0 / 3.0;
#ifdef MHD
  size_t const n_fields = 8;
#else   // not MHD
  size_t const n_fields = 5;
#endif  // MHD

  // Setup host grid. Fill host grid with random values and randomly assign maximum value
  std::vector<double> host_grid(nx * ny * nz * n_fields);
  for (double &val : host_grid) {
    val = doubleRand(prng);
  }

  // Allocating and copying to device
  cuda_utilities::DeviceVector<double> dev_grid(host_grid.size());
  dev_grid.cpyHostToDevice(host_grid);

// Fiducial Data
#ifdef MHD
  #ifdef PPMC
  std::vector<std::unordered_map<int, double>> fiducial_interface_left = {{{86, 3.6926886385390683},
                                                                           {302, 2.3022467009220993},
                                                                           {518, 2.3207781368125389},
                                                                           {734, 2.6544338753333747},
                                                                           {950, 11.430630157120799},
                                                                           {1166, 0.6428577630032507},
                                                                           {1382, 4.1406925096276597}},
                                                                          {{86, 3.811691682348938},
                                                                           {302, 1.4827993897794758},
                                                                           {518, 2.3955690789476871},
                                                                           {734, 4.06241130448349},
                                                                           {950, 10.552876853630949},
                                                                           {1166, 3.5147238706385471},
                                                                           {1382, 1.2344879085821312}},
                                                                          {{86, 3.1608655959160155},
                                                                           {302, 1.5377824007725194},
                                                                           {518, 0.41798730655927896},
                                                                           {734, 2.2721408530383784},
                                                                           {950, 5.6329522765789646},
                                                                           {1166, 0.84450832590555991},
                                                                           {1382, 1.4279317910797107}}};

  std::vector<std::unordered_map<int, double>> fiducial_interface_right = {{{85, 2.8949509658187838},
                                                                            {301, 0.25766140043685887},
                                                                            {517, 1.8194165731976308},
                                                                            {733, 2.0809921071868756},
                                                                            {949, 8.1315538869542046},
                                                                            {1165, 0.49708185787322312},
                                                                            {1381, 3.2017395511439881}},
                                                                           {{80, 2.8600082827930269},
                                                                            {296, 0.37343415089084014},
                                                                            {512, 1.7974558224423689},
                                                                            {728, 0.94369445956099784},
                                                                            {944, 7.7011501503138504},
                                                                            {1160, 3.5147238706385471},
                                                                            {1376, 1.2344879085821312}},
                                                                           {{50, 3.1608655959160155},
                                                                            {266, 0.32035830490636008},
                                                                            {482, 3.1721881746709815},
                                                                            {698, 2.2721408530383784},
                                                                            {914, 14.017699282483312},
                                                                            {1130, 1.5292690020097823},
                                                                            {1346, -0.12121484974901264}}};
  #else   // PPMC
  std::vector<std::unordered_map<int, double>> fiducial_interface_left = {{{86, 3.1608646282711232},
                                                                           {302, 0.84444422521258167},
                                                                           {518, 1.2459789393105685},
                                                                           {734, 2.2721401574613527},
                                                                           {950, 7.7508629541568022},
                                                                           {1166, 0.54567382624989913},
                                                                           {1382, 3.5147238706385462}},
                                                                          {{86, 3.6292858956631076},
                                                                           {302, 1.8316886259802778},
                                                                           {518, 2.2809308293670103},
                                                                           {734, 3.6939841768696002},
                                                                           {950, 10.405768833830281},
                                                                           {1166, 3.5147238706385462},
                                                                           {1382, 1.234487908582131}},
                                                                          {{86, 3.1608646282711232},
                                                                           {302, 0.84444422521258167},
                                                                           {518, 1.9865377887960551},
                                                                           {734, 1.1540870822905045},
                                                                           {950, 4.8971025794015812},
                                                                           {1166, 1.234487908582131},
                                                                           {1382, 0.54567382624989913}}};

  std::vector<std::unordered_map<int, double>> fiducial_interface_right = {{{301, 0.84444422521258167},
                                                                            {85, 3.1608646282711232},
                                                                            {733, 2.2721401574613527},
                                                                            {517, 3.2701799807980008},
                                                                            {949, 10.497902459040514},
                                                                            {1165, 0.54567382624989913},
                                                                            {1381, 3.5147238706385462}},
                                                                           {{80, 2.245959460360242},
                                                                            {296, 0.33326844362749702},
                                                                            {512, 1.4115388872411132},
                                                                            {728, 0.72702830835784316},
                                                                            {944, 7.5422056995631559},
                                                                            {1160, 3.5147238706385462},
                                                                            {1376, 1.234487908582131}},
                                                                           {{50, 3.1608646282711232},
                                                                            {266, 0.84444422521258167},
                                                                            {482, 1.9865377887960551},
                                                                            {698, 4.1768690252280765},
                                                                            {914, 14.823997016980297},
                                                                            {1130, 1.234487908582131},
                                                                            {1346, 0.54567382624989913}}};
  #endif  // PPMC
#else     // not MHD
  #ifdef PPMC
  std::vector<std::unordered_map<int, double>> fiducial_interface_left = {
      {{86, 4.155160222900312}, {302, 1.1624633361407897}, {518, 1.6379195998743412}, {734, 2.9868746414179093}},
      {{86, 4.1795874335665655}, {302, 2.1094239978455054}, {518, 2.6811988240843849}, {734, 4.2540957888954054}},
      {{86, 2.1772852940944429}, {302, 0.58167501916840214}, {518, 1.3683785996473696}, {734, 0.40276763592716164}}};

  std::vector<std::unordered_map<int, double>> fiducial_interface_right = {{{54, 3.8655260187947502},
                                                                            {85, 2.6637168309565289},
                                                                            {301, 0.69483650107094164},
                                                                            {517, 2.7558388224532218},
                                                                            {733, 1.9147729154830744}},
                                                                           {{54, 5.7556871317935459},
                                                                            {80, 2.6515032256234021},
                                                                            {296, 0.39344537106429511},
                                                                            {512, 1.6491544916805785},
                                                                            {728, 0.85830485311660487}},
                                                                           {{50, 2.8254070932730269},
                                                                            {54, 2.1884721760267873},
                                                                            {266, 0.75482470285166003},
                                                                            {482, 1.7757096932649317},
                                                                            {698, 3.6101832818706452}}};
  #else   // PPMC
  std::vector<std::unordered_map<int, double>> fiducial_interface_left = {
      {{86, 3.1608646282711232}, {302, 0.84444422521258167}, {518, 1.2459789393105685}, {734, 2.2721401574613527}},
      {{86, 3.6292858956631076}, {302, 1.8316886259802778}, {518, 2.2809308293670103}, {734, 3.6939841768696002}},
      {{86, 3.1608646282711232}, {302, 0.84444422521258167}, {518, 1.9865377887960551}, {734, 1.1540870822905045}}};

  std::vector<std::unordered_map<int, double>> fiducial_interface_right = {{{54, 3.4283787020401455},
                                                                            {85, 3.1608646282711232},
                                                                            {301, 0.84444422521258167},
                                                                            {517, 3.2701799807980008},
                                                                            {733, 2.2721401574613527}},
                                                                           {{54, 5.3122571267813665},
                                                                            {80, 2.245959460360242},
                                                                            {296, 0.33326844362749702},
                                                                            {512, 1.4115388872411132},
                                                                            {728, 0.72702830835784316}},
                                                                           {{50, 3.1608646282711232},
                                                                            {54, 3.2010935757366896},
                                                                            {266, 0.84444422521258167},
                                                                            {482, 1.9865377887960551},
                                                                            {698, 4.1768690252280765}}};
  #endif  // PPMC
#endif    // MHD

  // Loop over different directions
  for (size_t direction = 0; direction < 3; direction++) {
    // Allocate device buffers
    cuda_utilities::DeviceVector<double> dev_interface_left(nx * ny * nz * (n_fields - 1), true);
    cuda_utilities::DeviceVector<double> dev_interface_right(nx * ny * nz * (n_fields - 1), true);

    // Launch kernel
    switch (direction) {
      case 0:
        hipLaunchKernelGGL(PPM_cuda<0>, dev_grid.size(), 1, 0, 0, dev_grid.data(), dev_interface_left.data(),
                           dev_interface_right.data(), nx, ny, nz, 0, 0, gamma);
        break;
      case 1:
        hipLaunchKernelGGL(PPM_cuda<1>, dev_grid.size(), 1, 0, 0, dev_grid.data(), dev_interface_left.data(),
                           dev_interface_right.data(), nx, ny, nz, 0, 0, gamma);
        break;
      case 2:
        hipLaunchKernelGGL(PPM_cuda<2>, dev_grid.size(), 1, 0, 0, dev_grid.data(), dev_interface_left.data(),
                           dev_interface_right.data(), nx, ny, nz, 0, 0, gamma);
        break;
    }
    GPU_Error_Check();
    GPU_Error_Check(hipDeviceSynchronize());

    // Perform Comparison
    for (size_t i = 0; i < dev_interface_left.size(); i++) {
      // Check the left interface
      double test_val = dev_interface_left.at(i);
      double fiducial_val =
          (fiducial_interface_left.at(direction).find(i) == fiducial_interface_left.at(direction).end())
              ? 0.0
              : fiducial_interface_left.at(direction)[i];

      testing_utilities::Check_Results(
          fiducial_val, test_val,
          "left interface at i=" + std::to_string(i) + ", in direction " + std::to_string(direction));

      // Check the right interface
      test_val     = dev_interface_right.at(i);
      fiducial_val = (fiducial_interface_right.at(direction).find(i) == fiducial_interface_right.at(direction).end())
                         ? 0.0
                         : fiducial_interface_right.at(direction)[i];

      testing_utilities::Check_Results(
          fiducial_val, test_val,
          "right interface at i=" + std::to_string(i) + ", in direction " + std::to_string(direction));
    }
  }
}
