#include "hip/hip_runtime.h"
/*! \file ppm_cuda.cu
 *  \brief Functions definitions for the ppm kernels, using characteristic
 tracing. Written following Stone et al. 2008. */

#include <math.h>

#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../reconstruction/ppm_cuda.h"
#include "../reconstruction/reconstruction_internals.h"
#include "../utils/gpu.hpp"
#include "../utils/hydro_utilities.h"

#ifdef DE  // PRESSURE_DE
  #include "../utils/hydro_utilities.h"
#endif

// =====================================================================================================================
template <int dir>
__global__ __launch_bounds__(TPB) void PPM_cuda(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R, int nx,
                                                int ny, int nz, Real dx, Real dt, Real gamma)
{
  // get a thread ID
  int const thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  int xid, yid, zid;
  cuda_utilities::compute3DIndices(thread_id, nx, ny, xid, yid, zid);

  // Ensure that we are only operating on cells that will be used
  if (reconstruction::Thread_Guard<3>(nx, ny, nz, xid, yid, zid)) {
    return;
  }

  // Compute the total number of cells
  int const n_cells = nx * ny * nz;

  // Set the field indices for the various directions
  int o1, o2, o3;
  if constexpr (dir == 0) {
    o1 = grid_enum::momentum_x;
    o2 = grid_enum::momentum_y;
    o3 = grid_enum::momentum_z;
  } else if constexpr (dir == 1) {
    o1 = grid_enum::momentum_y;
    o2 = grid_enum::momentum_z;
    o3 = grid_enum::momentum_x;
  } else if constexpr (dir == 2) {
    o1 = grid_enum::momentum_z;
    o2 = grid_enum::momentum_x;
    o3 = grid_enum::momentum_y;
  }

  // load the 5-cell stencil into registers
  // cell i
  hydro_utilities::Primitive const cell_i =
      hydro_utilities::Load_Cell_Primitive<dir>(dev_conserved, xid, yid, zid, nx, ny, n_cells, gamma);

  // cell i-1. The equality checks the direction and will subtract one from the correct direction
  // im1 stands for "i minus 1"
  hydro_utilities::Primitive const cell_im1 = hydro_utilities::Load_Cell_Primitive<dir>(
      dev_conserved, xid - int(dir == 0), yid - int(dir == 1), zid - int(dir == 2), nx, ny, n_cells, gamma);

  // cell i+1.  The equality checks the direction and add one to the correct direction
  // ip1 stands for "i plus 1"
  hydro_utilities::Primitive const cell_ip1 = hydro_utilities::Load_Cell_Primitive<dir>(
      dev_conserved, xid + int(dir == 0), yid + int(dir == 1), zid + int(dir == 2), nx, ny, n_cells, gamma);

  // cell i-2. The equality checks the direction and will subtract two from the correct direction
  // im2 stands for "i minus 2"
  hydro_utilities::Primitive const cell_im2 = hydro_utilities::Load_Cell_Primitive<dir>(
      dev_conserved, xid - 2 * int(dir == 0), yid - 2 * int(dir == 1), zid - 2 * int(dir == 2), nx, ny, n_cells, gamma);

  // cell i+2.  The equality checks the direction and add two to the correct direction
  // ip2 stands for "i plus 2"
  hydro_utilities::Primitive const cell_ip2 = hydro_utilities::Load_Cell_Primitive<dir>(
      dev_conserved, xid + 2 * int(dir == 0), yid + 2 * int(dir == 1), zid + 2 * int(dir == 2), nx, ny, n_cells, gamma);

#ifdef PPMC
  // Compute the eigenvectors
  reconstruction::EigenVecs const eigenvectors = reconstruction::Compute_Eigenvectors(cell_i, gamma);

  // Cell i
  reconstruction::Characteristic const cell_i_characteristic =
      reconstruction::Primitive_To_Characteristic(cell_i, cell_i, eigenvectors, gamma);

  // Cell i-1
  reconstruction::Characteristic const cell_im1_characteristic =
      reconstruction::Primitive_To_Characteristic(cell_i, cell_im1, eigenvectors, gamma);

  // Cell i-2
  reconstruction::Characteristic const cell_im2_characteristic =
      reconstruction::Primitive_To_Characteristic(cell_i, cell_im2, eigenvectors, gamma);

  // Cell i+1
  reconstruction::Characteristic const cell_ip1_characteristic =
      reconstruction::Primitive_To_Characteristic(cell_i, cell_ip1, eigenvectors, gamma);

  // Cell i+2
  reconstruction::Characteristic const cell_ip2_characteristic =
      reconstruction::Primitive_To_Characteristic(cell_i, cell_ip2, eigenvectors, gamma);

  // Compute the interface states for each field
  auto const [interface_L_iph_characteristic, interface_R_imh_characteristic] =
      reconstruction::PPM_Interfaces(cell_im2_characteristic, cell_im1_characteristic, cell_i_characteristic,
                                     cell_ip1_characteristic, cell_ip2_characteristic);

  // Convert back to primitive variables
  hydro_utilities::Primitive interface_L_iph =
      reconstruction::Characteristic_To_Primitive(cell_i, interface_L_iph_characteristic, eigenvectors, gamma);
  hydro_utilities::Primitive interface_R_imh =
      reconstruction::Characteristic_To_Primitive(cell_i, interface_R_imh_characteristic, eigenvectors, gamma);

  // Compute the interfaces for the variables that don't have characteristics
  #ifdef DE
  reconstruction::PPM_Single_Variable(cell_im2.gas_energy_specific, cell_im1.gas_energy_specific,
                                      cell_i.gas_energy_specific, cell_ip1.gas_energy_specific,
                                      cell_ip2.gas_energy_specific, interface_L_iph.gas_energy_specific,
                                      interface_R_imh.gas_energy_specific);
  #endif  // DE
  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    reconstruction::PPM_Single_Variable(cell_im2.scalar_specific[i], cell_im1.scalar_specific[i],
                                        cell_i.scalar_specific[i], cell_ip1.scalar_specific[i],
                                        cell_ip2.scalar_specific[i], interface_L_iph.scalar_specific[i],
                                        interface_R_imh.scalar_specific[i]);
  }
  #endif  // SCALAR
#else     // PPMC
  auto [interface_L_iph, interface_R_imh] =
      reconstruction::PPM_Interfaces(cell_im2, cell_im1, cell_i, cell_ip1, cell_ip2);
#endif    // PPMC

  // Do the characteristic tracing
#ifndef VL
  PPM_Characteristic_Evolution(cell_i, dt, dx, gamma, interface_R_imh, interface_L_iph);
#endif  // VL

  // enforce minimum values
  interface_R_imh.density  = fmax(interface_R_imh.density, (Real)TINY_NUMBER);
  interface_L_iph.density  = fmax(interface_L_iph.density, (Real)TINY_NUMBER);
  interface_R_imh.pressure = fmax(interface_R_imh.pressure, (Real)TINY_NUMBER);
  interface_L_iph.pressure = fmax(interface_L_iph.pressure, (Real)TINY_NUMBER);

  // Step 11 - Send final values back from kernel

  // Convert the left and right states in the primitive to the conserved variables send final values back from kernel
  // bounds_R refers to the right side of the i-1/2 interface
  size_t id = cuda_utilities::compute1DIndex(xid, yid, zid, nx, ny);
  reconstruction::Write_Data(interface_L_iph, dev_bounds_L, dev_conserved, id, n_cells, o1, o2, o3, gamma);

  id = cuda_utilities::compute1DIndex(xid - int(dir == 0), yid - int(dir == 1), zid - int(dir == 2), nx, ny);
  reconstruction::Write_Data(interface_R_imh, dev_bounds_R, dev_conserved, id, n_cells, o1, o2, o3, gamma);
}
// Instantiate the relevant template specifications
template __global__ __launch_bounds__(TPB) void PPM_cuda<0>(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R,
                                                            int nx, int ny, int nz, Real dx, Real dt, Real gamma);
template __global__ __launch_bounds__(TPB) void PPM_cuda<1>(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R,
                                                            int nx, int ny, int nz, Real dx, Real dt, Real gamma);
template __global__ __launch_bounds__(TPB) void PPM_cuda<2>(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R,
                                                            int nx, int ny, int nz, Real dx, Real dt, Real gamma);
// =====================================================================================================================
