#include "hip/hip_runtime.h"
/*! \file ppm_cuda.cu
 *  \brief Functions definitions for the ppm kernels, using characteristic
 tracing. Written following Stone et al. 2008. */

#include <math.h>

#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../reconstruction/ppm_cuda.h"
#include "../reconstruction/reconstruction_internals.h"
#include "../utils/gpu.hpp"
#include "../utils/hydro_utilities.h"

#ifdef DE  // PRESSURE_DE
  #include "../utils/hydro_utilities.h"
#endif

void __device__ __host__ PPM_Characteristic_Evolution(hydro_utilities::Primitive const cell_i, Real const dt,
                                                      Real const dx, Real const gamma,
                                                      hydro_utilities::Primitive &interface_R_imh,
                                                      hydro_utilities::Primitive &interface_L_iph)
{
  // This is the beginning of the characteristic tracing
  // Step 8 - Compute the coefficients for the monotonized parabolic
  // interpolation function
  //          Stone Eqn 54
  hydro_utilities::Primitive interface_slope;
  interface_slope.density      = interface_L_iph.density - interface_R_imh.density;
  interface_slope.velocity.x() = interface_L_iph.velocity.x() - interface_R_imh.velocity.x();
  interface_slope.velocity.y() = interface_L_iph.velocity.y() - interface_R_imh.velocity.y();
  interface_slope.velocity.z() = interface_L_iph.velocity.z() - interface_R_imh.velocity.z();
  interface_slope.pressure     = interface_L_iph.pressure - interface_R_imh.pressure;

  Real const d_6  = 6.0 * (cell_i.density - 0.5 * (interface_R_imh.density + interface_L_iph.density));
  Real const vx_6 = 6.0 * (cell_i.velocity.x() - 0.5 * (interface_R_imh.velocity.x() + interface_L_iph.velocity.x()));
  Real const vy_6 = 6.0 * (cell_i.velocity.y() - 0.5 * (interface_R_imh.velocity.y() + interface_L_iph.velocity.y()));
  Real const vz_6 = 6.0 * (cell_i.velocity.z() - 0.5 * (interface_R_imh.velocity.z() + interface_L_iph.velocity.z()));
  Real const p_6  = 6.0 * (cell_i.pressure - 0.5 * (interface_R_imh.pressure + interface_L_iph.pressure));

#ifdef DE
  interface_slope.gas_energy_specific = interface_L_iph.gas_energy_specific - interface_R_imh.gas_energy_specific;
  Real const ge_6                     = 6.0 * (cell_i.gas_energy_specific -
                           0.5 * (interface_R_imh.gas_energy_specific + interface_L_iph.gas_energy_specific));
#endif  // DE

#ifdef SCALAR
  Real scalar_6[NSCALARS];
  for (int i = 0; i < NSCALARS; i++) {
    interface_slope.scalar_specific[i] = interface_L_iph.scalar_specific[i] - interface_R_imh.scalar_specific[i];
    scalar_6[i]                        = 6.0 * (cell_i.scalar_specific[i] -
                         0.5 * (interface_R_imh.scalar_specific[i] + interface_L_iph.scalar_specific[i]));
  }
#endif  // SCALAR

  // Compute the eigenvalues of the linearized equations in the
  // primitive variables using the cell-centered primitive variables

  // recalculate the adiabatic sound speed in cell i
  Real const sound_speed = hydro_utilities::Calc_Sound_Speed(cell_i.pressure, cell_i.density, gamma);

  Real const lambda_m = cell_i.velocity.x() - sound_speed;
  Real const lambda_0 = cell_i.velocity.x();
  Real const lambda_p = cell_i.velocity.x() + sound_speed;

  // Step 9 - Compute the left and right interface values using monotonized
  // parabolic interpolation
  //          Stone Eqns 55 & 56

  // largest eigenvalue
  Real const lambda_max = fmax(lambda_p, (Real)0);
  // smallest eigenvalue
  Real const lambda_min = fmin(lambda_m, (Real)0);

  // left interface value, i+1/2
  Real const dtodx = dt / dx;
  interface_L_iph.density =
      interface_L_iph.density -
      lambda_max * (0.5 * dtodx) * (interface_slope.density - (1.0 - (2.0 / 3.0) * lambda_max * dtodx) * d_6);
  interface_L_iph.velocity.x() =
      interface_L_iph.velocity.x() -
      lambda_max * (0.5 * dtodx) * (interface_slope.velocity.x() - (1.0 - (2.0 / 3.0) * lambda_max * dtodx) * vx_6);
  interface_L_iph.velocity.y() =
      interface_L_iph.velocity.y() -
      lambda_max * (0.5 * dtodx) * (interface_slope.velocity.y() - (1.0 - (2.0 / 3.0) * lambda_max * dtodx) * vy_6);
  interface_L_iph.velocity.z() =
      interface_L_iph.velocity.z() -
      lambda_max * (0.5 * dtodx) * (interface_slope.velocity.z() - (1.0 - (2.0 / 3.0) * lambda_max * dtodx) * vz_6);
  interface_L_iph.pressure =
      interface_L_iph.pressure -
      lambda_max * (0.5 * dtodx) * (interface_slope.pressure - (1.0 - (2.0 / 3.0) * lambda_max * dtodx) * p_6);

  // right interface value, i-1/2
  interface_R_imh.density =
      interface_R_imh.density -
      lambda_min * (0.5 * dtodx) * (interface_slope.density + (1.0 + (2.0 / 3.0) * lambda_min * dtodx) * d_6);
  interface_R_imh.velocity.x() =
      interface_R_imh.velocity.x() -
      lambda_min * (0.5 * dtodx) * (interface_slope.velocity.x() + (1.0 + (2.0 / 3.0) * lambda_min * dtodx) * vx_6);
  interface_R_imh.velocity.y() =
      interface_R_imh.velocity.y() -
      lambda_min * (0.5 * dtodx) * (interface_slope.velocity.y() + (1.0 + (2.0 / 3.0) * lambda_min * dtodx) * vy_6);
  interface_R_imh.velocity.z() =
      interface_R_imh.velocity.z() -
      lambda_min * (0.5 * dtodx) * (interface_slope.velocity.z() + (1.0 + (2.0 / 3.0) * lambda_min * dtodx) * vz_6);
  interface_R_imh.pressure =
      interface_R_imh.pressure -
      lambda_min * (0.5 * dtodx) * (interface_slope.pressure + (1.0 + (2.0 / 3.0) * lambda_min * dtodx) * p_6);

#ifdef DE
  interface_L_iph.gas_energy_specific =
      interface_L_iph.gas_energy_specific -
      lambda_max * (0.5 * dtodx) *
          (interface_slope.gas_energy_specific - (1.0 - (2.0 / 3.0) * lambda_max * dtodx) * ge_6);
  interface_R_imh.gas_energy_specific =
      interface_R_imh.gas_energy_specific -
      lambda_min * (0.5 * dtodx) *
          (interface_slope.gas_energy_specific + (1.0 + (2.0 / 3.0) * lambda_min * dtodx) * ge_6);
#endif  // DE

#ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    interface_L_iph.scalar_specific[i] =
        interface_L_iph.scalar_specific[i] -
        lambda_max * (0.5 * dtodx) *
            (interface_slope.scalar_specific[i] - (1.0 - (2.0 / 3.0) * lambda_max * dtodx) * scalar_6[i]);
    interface_R_imh.scalar_specific[i] =
        interface_R_imh.scalar_specific[i] -
        lambda_min * (0.5 * dtodx) *
            (interface_slope.scalar_specific[i] + (1.0 + (2.0 / 3.0) * lambda_min * dtodx) * scalar_6[i]);
  }
#endif  // SCALAR

  // Step 10 - Perform the characteristic tracing
  //           Stone Eqns 57 - 60

  // left-hand interface value, i+1/2
  Real sum_1 = 0, sum_2 = 0, sum_3 = 0, sum_4 = 0, sum_5 = 0;
#ifdef DE
  Real sum_ge = 0;
  Real chi_ge = 0;
#endif  // DE
#ifdef SCALAR
  Real chi_scalar[NSCALARS];
  Real sum_scalar[NSCALARS];
  for (Real &val : sum_scalar) {
    val = 0;
  }
#endif  // SCALAR

  if (lambda_m >= 0) {
    Real const A = (0.5 * dtodx) * (lambda_p - lambda_m);
    Real const B = (1.0 / 3.0) * (dtodx) * (dtodx) * (lambda_p * lambda_p - lambda_m * lambda_m);

    Real const chi_1 = A * (interface_slope.density - d_6) + B * d_6;
    Real const chi_2 = A * (interface_slope.velocity.x() - vx_6) + B * vx_6;
    Real const chi_3 = A * (interface_slope.velocity.y() - vy_6) + B * vy_6;
    Real const chi_4 = A * (interface_slope.velocity.z() - vz_6) + B * vz_6;
    Real const chi_5 = A * (interface_slope.pressure - p_6) + B * p_6;

    sum_1 += -0.5 * (cell_i.density * chi_2 / sound_speed - chi_5 / (sound_speed * sound_speed));
    sum_2 += 0.5 * (chi_2 - chi_5 / (sound_speed * cell_i.density));
    sum_5 += -0.5 * (cell_i.density * chi_2 * sound_speed - chi_5);
  }
  if (lambda_0 >= 0) {
    Real const A = (0.5 * dtodx) * (lambda_p - lambda_0);
    Real const B = (1.0 / 3.0) * (dtodx) * (dtodx) * (lambda_p * lambda_p - lambda_0 * lambda_0);

    Real const chi_1 = A * (interface_slope.density - d_6) + B * d_6;
    Real const chi_2 = A * (interface_slope.velocity.x() - vx_6) + B * vx_6;
    Real const chi_3 = A * (interface_slope.velocity.y() - vy_6) + B * vy_6;
    Real const chi_4 = A * (interface_slope.velocity.z() - vz_6) + B * vz_6;
    Real const chi_5 = A * (interface_slope.pressure - p_6) + B * p_6;
#ifdef DE
    chi_ge = A * (interface_slope.gas_energy_specific - ge_6) + B * ge_6;
#endif  // DE
#ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      chi_scalar[i] = A * (interface_slope.scalar_specific[i] - scalar_6[i]) + B * scalar_6[i];
    }
#endif  // SCALAR

    sum_1 += chi_1 - chi_5 / (sound_speed * sound_speed);
    sum_3 += chi_3;
    sum_4 += chi_4;
#ifdef DE
    sum_ge += chi_ge;
#endif  // DE
#ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      sum_scalar[i] += chi_scalar[i];
    }
#endif  // SCALAR
  }
  if (lambda_p >= 0) {
    Real const A = (0.5 * dtodx) * (lambda_p - lambda_p);
    Real const B = (1.0 / 3.0) * (dtodx) * (dtodx) * (lambda_p * lambda_p - lambda_p * lambda_p);

    Real const chi_1 = A * (interface_slope.density - d_6) + B * d_6;
    Real const chi_2 = A * (interface_slope.velocity.x() - vx_6) + B * vx_6;
    Real const chi_3 = A * (interface_slope.velocity.y() - vy_6) + B * vy_6;
    Real const chi_4 = A * (interface_slope.velocity.z() - vz_6) + B * vz_6;
    Real const chi_5 = A * (interface_slope.pressure - p_6) + B * p_6;

    sum_1 += 0.5 * (cell_i.density * chi_2 / sound_speed + chi_5 / (sound_speed * sound_speed));
    sum_2 += 0.5 * (chi_2 + chi_5 / (sound_speed * cell_i.density));
    sum_5 += 0.5 * (cell_i.density * chi_2 * sound_speed + chi_5);
  }

  // add the corrections to the initial guesses for the interface values
  interface_L_iph.density += sum_1;
  interface_L_iph.velocity.x() += sum_2;
  interface_L_iph.velocity.y() += sum_3;
  interface_L_iph.velocity.z() += sum_4;
  interface_L_iph.pressure += sum_5;
#ifdef DE
  interface_L_iph.gas_energy_specific += sum_ge;
#endif  // DE
#ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    interface_L_iph.scalar_specific[i] += sum_scalar[i];
  }
#endif  // SCALAR

  // right-hand interface value, i-1/2
  sum_1 = 0;
  sum_2 = 0;
  sum_3 = 0;
  sum_4 = 0;
  sum_5 = 0;
#ifdef DE
  sum_ge = 0;
#endif  // DE
#ifdef SCALAR
  for (Real &val : sum_scalar) {
    val = 0;
  }
#endif  // SCALAR
  if (lambda_m <= 0) {
    Real const C = (0.5 * dtodx) * (lambda_m - lambda_m);
    Real const D = (1.0 / 3.0) * (dtodx) * (dtodx) * (lambda_m * lambda_m - lambda_m * lambda_m);

    Real const chi_1 = C * (interface_slope.density + d_6) + D * d_6;
    Real const chi_2 = C * (interface_slope.velocity.x() + vx_6) + D * vx_6;
    Real const chi_3 = C * (interface_slope.velocity.y() + vy_6) + D * vy_6;
    Real const chi_4 = C * (interface_slope.velocity.z() + vz_6) + D * vz_6;
    Real const chi_5 = C * (interface_slope.pressure + p_6) + D * p_6;

    sum_1 += -0.5 * (cell_i.density * chi_2 / sound_speed - chi_5 / (sound_speed * sound_speed));
    sum_2 += 0.5 * (chi_2 - chi_5 / (sound_speed * cell_i.density));
    sum_5 += -0.5 * (cell_i.density * chi_2 * sound_speed - chi_5);
  }
  if (lambda_0 <= 0) {
    Real const C = (0.5 * dtodx) * (lambda_m - lambda_0);
    Real const D = (1.0 / 3.0) * (dtodx) * (dtodx) * (lambda_m * lambda_m - lambda_0 * lambda_0);

    Real const chi_1 = C * (interface_slope.density + d_6) + D * d_6;
    Real const chi_2 = C * (interface_slope.velocity.x() + vx_6) + D * vx_6;
    Real const chi_3 = C * (interface_slope.velocity.y() + vy_6) + D * vy_6;
    Real const chi_4 = C * (interface_slope.velocity.z() + vz_6) + D * vz_6;
    Real const chi_5 = C * (interface_slope.pressure + p_6) + D * p_6;
#ifdef DE
    chi_ge = C * (interface_slope.gas_energy_specific + ge_6) + D * ge_6;
#endif  // DE
#ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      chi_scalar[i] = C * (interface_slope.scalar_specific[i] + scalar_6[i]) + D * scalar_6[i];
    }
#endif  // SCALAR

    sum_1 += chi_1 - chi_5 / (sound_speed * sound_speed);
    sum_3 += chi_3;
    sum_4 += chi_4;
#ifdef DE
    sum_ge += chi_ge;
#endif  // DE
#ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      sum_scalar[i] += chi_scalar[i];
    }
#endif  // SCALAR
  }
  if (lambda_p <= 0) {
    Real const C = (0.5 * dtodx) * (lambda_m - lambda_p);
    Real const D = (1.0 / 3.0) * (dtodx) * (dtodx) * (lambda_m * lambda_m - lambda_p * lambda_p);

    Real const chi_1 = C * (interface_slope.density + d_6) + D * d_6;
    Real const chi_2 = C * (interface_slope.velocity.x() + vx_6) + D * vx_6;
    Real const chi_3 = C * (interface_slope.velocity.y() + vy_6) + D * vy_6;
    Real const chi_4 = C * (interface_slope.velocity.z() + vz_6) + D * vz_6;
    Real const chi_5 = C * (interface_slope.pressure + p_6) + D * p_6;

    sum_1 += 0.5 * (cell_i.density * chi_2 / sound_speed + chi_5 / (sound_speed * sound_speed));
    sum_2 += 0.5 * (chi_2 + chi_5 / (sound_speed * cell_i.density));
    sum_5 += 0.5 * (cell_i.density * chi_2 * sound_speed + chi_5);
  }

  // add the corrections
  interface_R_imh.density += sum_1;
  interface_R_imh.velocity.x() += sum_2;
  interface_R_imh.velocity.y() += sum_3;
  interface_R_imh.velocity.z() += sum_4;
  interface_R_imh.pressure += sum_5;
#ifdef DE
  interface_R_imh.gas_energy_specific += sum_ge;
#endif  // DE
#ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    interface_R_imh.scalar_specific[i] += sum_scalar[i];
  }
#endif  // SCALAR

  // This is the end of the characteristic tracing
}
// =====================================================================================================================
template <int dir>
__global__ __launch_bounds__(TPB) void PPM_cuda(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R, int nx,
                                                int ny, int nz, Real dx, Real dt, Real gamma)
{
  // get a thread ID
  int const thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  int xid, yid, zid;
  cuda_utilities::compute3DIndices(thread_id, nx, ny, xid, yid, zid);

  // Ensure that we are only operating on cells that will be used
  if (reconstruction::Thread_Guard<3>(nx, ny, nz, xid, yid, zid)) {
    return;
  }

  // Compute the total number of cells
  int const n_cells = nx * ny * nz;

  // Set the field indices for the various directions
  int o1, o2, o3;
  if constexpr (dir == 0) {
    o1 = grid_enum::momentum_x;
    o2 = grid_enum::momentum_y;
    o3 = grid_enum::momentum_z;
  } else if constexpr (dir == 1) {
    o1 = grid_enum::momentum_y;
    o2 = grid_enum::momentum_z;
    o3 = grid_enum::momentum_x;
  } else if constexpr (dir == 2) {
    o1 = grid_enum::momentum_z;
    o2 = grid_enum::momentum_x;
    o3 = grid_enum::momentum_y;
  }

  // load the 5-cell stencil into registers
  // cell i
  hydro_utilities::Primitive const cell_i =
      hydro_utilities::Load_Cell_Primitive<dir>(dev_conserved, xid, yid, zid, nx, ny, n_cells, gamma);

  // cell i-1. The equality checks the direction and will subtract one from the correct direction
  // im1 stands for "i minus 1"
  hydro_utilities::Primitive const cell_im1 = hydro_utilities::Load_Cell_Primitive<dir>(
      dev_conserved, xid - int(dir == 0), yid - int(dir == 1), zid - int(dir == 2), nx, ny, n_cells, gamma);

  // cell i+1.  The equality checks the direction and add one to the correct direction
  // ip1 stands for "i plus 1"
  hydro_utilities::Primitive const cell_ip1 = hydro_utilities::Load_Cell_Primitive<dir>(
      dev_conserved, xid + int(dir == 0), yid + int(dir == 1), zid + int(dir == 2), nx, ny, n_cells, gamma);

  // cell i-2. The equality checks the direction and will subtract two from the correct direction
  // im2 stands for "i minus 2"
  hydro_utilities::Primitive const cell_im2 = hydro_utilities::Load_Cell_Primitive<dir>(
      dev_conserved, xid - 2 * int(dir == 0), yid - 2 * int(dir == 1), zid - 2 * int(dir == 2), nx, ny, n_cells, gamma);

  // cell i+2.  The equality checks the direction and add two to the correct direction
  // ip2 stands for "i plus 2"
  hydro_utilities::Primitive const cell_ip2 = hydro_utilities::Load_Cell_Primitive<dir>(
      dev_conserved, xid + 2 * int(dir == 0), yid + 2 * int(dir == 1), zid + 2 * int(dir == 2), nx, ny, n_cells, gamma);

#ifdef PPMC
  // Compute the eigenvectors
  reconstruction::EigenVecs const eigenvectors = reconstruction::Compute_Eigenvectors(cell_i, gamma);

  // Cell i
  reconstruction::Characteristic const cell_i_characteristic =
      reconstruction::Primitive_To_Characteristic(cell_i, cell_i, eigenvectors, gamma);

  // Cell i-1
  reconstruction::Characteristic const cell_im1_characteristic =
      reconstruction::Primitive_To_Characteristic(cell_i, cell_im1, eigenvectors, gamma);

  // Cell i-2
  reconstruction::Characteristic const cell_im2_characteristic =
      reconstruction::Primitive_To_Characteristic(cell_i, cell_im2, eigenvectors, gamma);

  // Cell i+1
  reconstruction::Characteristic const cell_ip1_characteristic =
      reconstruction::Primitive_To_Characteristic(cell_i, cell_ip1, eigenvectors, gamma);

  // Cell i+2
  reconstruction::Characteristic const cell_ip2_characteristic =
      reconstruction::Primitive_To_Characteristic(cell_i, cell_ip2, eigenvectors, gamma);

  // Compute the interface states for each field
  auto const [interface_L_iph_characteristic, interface_R_imh_characteristic] =
      reconstruction::PPM_Interfaces(cell_im2_characteristic, cell_im1_characteristic, cell_i_characteristic,
                                     cell_ip1_characteristic, cell_ip2_characteristic);

  // Convert back to primitive variables
  hydro_utilities::Primitive interface_L_iph =
      reconstruction::Characteristic_To_Primitive(cell_i, interface_L_iph_characteristic, eigenvectors, gamma);
  hydro_utilities::Primitive interface_R_imh =
      reconstruction::Characteristic_To_Primitive(cell_i, interface_R_imh_characteristic, eigenvectors, gamma);

  // Compute the interfaces for the variables that don't have characteristics
  #ifdef DE
  reconstruction::PPM_Single_Variable(cell_im2.gas_energy_specific, cell_im1.gas_energy_specific,
                                      cell_i.gas_energy_specific, cell_ip1.gas_energy_specific,
                                      cell_ip2.gas_energy_specific, interface_L_iph.gas_energy_specific,
                                      interface_R_imh.gas_energy_specific);
  #endif  // DE
  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    reconstruction::PPM_Single_Variable(cell_im2.scalar_specific[i], cell_im1.scalar_specific[i],
                                        cell_i.scalar_specific[i], cell_ip1.scalar_specific[i],
                                        cell_ip2.scalar_specific[i], interface_L_iph.scalar_specific[i],
                                        interface_R_imh.scalar_specific[i]);
  }
  #endif  // SCALAR
#else     // PPMC
  auto [interface_L_iph, interface_R_imh] =
      reconstruction::PPM_Interfaces(cell_im2, cell_im1, cell_i, cell_ip1, cell_ip2);
#endif    // PPMC

  // Do the characteristic tracing
#ifndef VL
  PPM_Characteristic_Evolution(cell_i, dt, dx, gamma, interface_R_imh, interface_L_iph);
#endif  // VL

  // enforce minimum values
  interface_R_imh.density  = fmax(interface_R_imh.density, (Real)TINY_NUMBER);
  interface_L_iph.density  = fmax(interface_L_iph.density, (Real)TINY_NUMBER);
  interface_R_imh.pressure = fmax(interface_R_imh.pressure, (Real)TINY_NUMBER);
  interface_L_iph.pressure = fmax(interface_L_iph.pressure, (Real)TINY_NUMBER);

  // Step 11 - Send final values back from kernel

  // Convert the left and right states in the primitive to the conserved variables send final values back from kernel
  // bounds_R refers to the right side of the i-1/2 interface
  size_t id = cuda_utilities::compute1DIndex(xid, yid, zid, nx, ny);
  reconstruction::Write_Data(interface_L_iph, dev_bounds_L, dev_conserved, id, n_cells, o1, o2, o3, gamma);

  id = cuda_utilities::compute1DIndex(xid - int(dir == 0), yid - int(dir == 1), zid - int(dir == 2), nx, ny);
  reconstruction::Write_Data(interface_R_imh, dev_bounds_R, dev_conserved, id, n_cells, o1, o2, o3, gamma);
}
// Instantiate the relevant template specifications
template __global__ __launch_bounds__(TPB) void PPM_cuda<0>(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R,
                                                            int nx, int ny, int nz, Real dx, Real dt, Real gamma);
template __global__ __launch_bounds__(TPB) void PPM_cuda<1>(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R,
                                                            int nx, int ny, int nz, Real dx, Real dt, Real gamma);
template __global__ __launch_bounds__(TPB) void PPM_cuda<2>(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R,
                                                            int nx, int ny, int nz, Real dx, Real dt, Real gamma);
// =====================================================================================================================
